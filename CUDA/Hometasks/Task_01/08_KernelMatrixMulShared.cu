#include "hip/hip_runtime.h"
#include "08_KernelMatrixMulShared.cuh"

/* Произведение матриц (с использованием разделяемой памяти) */
__global__ 
void KernelMatrixMulShared(int m, int n, int p, const float* matrix_a, const float* matrix_b, float* result) {
    __shared__ float tile_a[TILE_DIM][TILE_DIM];
    __shared__ float tile_b[TILE_DIM][TILE_DIM];

    float sum = 0.0;

    for (int tile_idx = 0; tile_idx < ceilf(static_cast<float>(n) / TILE_DIM); ++tile_idx) {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = tile_idx * blockDim.x + threadIdx.x;

        if (row < m && col < n)
            tile_a[threadIdx.y][threadIdx.x] = matrix_a[row * n + col];
        else
            tile_a[threadIdx.y][threadIdx.x] = 0.0;

        row = tile_idx * blockDim.y + threadIdx.y;
        col = blockIdx.x * blockDim.x + threadIdx.x;

        if (row < n && col < p)
            tile_b[threadIdx.y][threadIdx.x] = matrix_b[row * p + col];
        else
            tile_b[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for (int k = 0; k < TILE_DIM; ++k)
            sum += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];

        __syncthreads();
    }

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < p)
        result[row * p + col] = sum;    
}

/* Произведение матриц, выполненное на хосте */
void MatrixMulHost(int m, int n, int p, const float* matrix_a, const float* matrix_b, float* result) {
    for (int i = 0; i < m; ++i)
        for (int j = 0; j < p; ++j)
            for (int k = 0; k < n; ++k)
                result[i * p + j] += matrix_a[i * n + k] * matrix_b[k * p + j];
}

/* Вывод матрицы на экран */
void PrintMatrix(int height, int width, const float* matrix) {
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j)
			std::cout << matrix[i * width + j] << " ";
        std::cout << std::endl;
	}
}

/* Проверка результата работы функции */
void CheckResults(int n, float* ref, float* result, float time_elapsed, const std::string& function_name) {
    for (int i = 0; i < n; ++i) {
        if (result[i] != ref[i]) {
            std::cout << "Failed test " << function_name << "!" << std::endl;
            return;
        }
    }
        
    std::cout << "Test " << function_name << " OK!" << std::endl;
    std::cout << "Time elapsed : " << time_elapsed / WARM_UP << " ms" << std::endl;
}

void TestKernelMatrixMulShared() {
    int rows_a = 1 << 10;
    int cols_a = 1 << 10;
    int rows_b = 1 << 10;
    int cols_b = 1 << 10;

    if (cols_a != rows_b) {
        std::cout << "The number of rows in matrix A should be equal to number of rows in matrix B" << std::endl;
        return;
    }

    int n_total_a = rows_a * cols_a;
    int n_total_b = rows_b * cols_b;
    int n_total_result = rows_a * cols_b;

    float* h_matrix_a = new float[n_total_a];
    float* h_matrix_b = new float[n_total_b];
    float* h_result = new float[n_total_result];
    float* check_result = new float[n_total_result];

    float *d_matrix_a, *d_matrix_b, *d_result;

    hipMalloc(&d_matrix_a, n_total_a * sizeof(float));
    hipMalloc(&d_matrix_b, n_total_b * sizeof(float));
    hipMalloc(&d_result, n_total_result * sizeof(float));

    dim3 threads_per_block(TILE_DIM, TILE_DIM);
    dim3 blocks_per_grid(1, 1);

    blocks_per_grid.x = std::ceil(static_cast<double>(cols_b) / static_cast<double>(threads_per_block.x));
    blocks_per_grid.y = std::ceil(static_cast<double>(rows_a) / static_cast<double>(threads_per_block.y));

    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(-256, 256);

    /* Инициализация матриц */
    for (int i = 0; i < rows_a; ++i)
        for (int j = 0; j < cols_a; ++j)
            h_matrix_a[i * cols_a + j] = static_cast<float>(uniform_dist(e));

    for (int i = 0; i < rows_b; ++i)
        for (int j = 0; j < cols_b; ++j)
            h_matrix_b[i * cols_b + j] = static_cast<float>(uniform_dist(e));

    MatrixMulHost(rows_a, cols_a, cols_b, h_matrix_a, h_matrix_b, check_result);

    hipMemcpy(d_matrix_a, h_matrix_a, n_total_a * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix_b, h_matrix_b, n_total_b * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time_elapsed;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* с использованием разделяемой памяти */
    {
        hipMemset(d_result, 0.0, n_total_result * sizeof(float));
        
        KernelMatrixMulShared<<<blocks_per_grid, threads_per_block>>>(rows_a, cols_a, cols_b, 
                d_matrix_a, d_matrix_b, d_result);

        hipEventRecord(start, 0);
        
        for (int i = 0; i < WARM_UP; ++i)
            KernelMatrixMulShared<<<blocks_per_grid, threads_per_block>>>(rows_a, cols_a, cols_b, 
                d_matrix_a, d_matrix_b, d_result);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop); 
        hipEventElapsedTime(&time_elapsed, start, stop);
        hipMemcpy(h_result, d_result, n_total_result * sizeof(float), hipMemcpyDeviceToHost);

        CheckResults(n_total_result, check_result, h_result, time_elapsed, "KernelMatrixMulShared");
    }

    hipFree(d_matrix_a);
    hipFree(d_matrix_b);
    hipFree(d_result);

    delete[] h_matrix_a;
    delete[] h_matrix_b;
    delete[] h_result;
    delete[] check_result;
}